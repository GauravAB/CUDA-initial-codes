#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include ""


__global__ void VecAdd(float *a ,float *b , float*c , int N)
{
	int idx = threadIdx.x;
	if (idx < N)
	{
		c[idx] = a[idx] + b[idx];
	}
}


void main(void)
{
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	const int ARRAY_SIZE = 1024;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
	
	float h_arrA[ARRAY_SIZE];
	float h_arrB[ARRAY_SIZE];
	float h_arrC[ARRAY_SIZE];
	float* d_arrA;
	float* d_arrB;
	float* d_arrC;

	int i;
	float f = 0.0;

	//initializing the host arrays
	for (i = 0; i < ARRAY_SIZE; i++)
	{
		h_arrA[i] = f;
		h_arrB[i] = f * f;
		f = f + 1.0;
	}
	//initializing the device arrays
	
	hipMalloc((void**)&d_arrA, ARRAY_BYTES);
	hipMalloc((void**)&d_arrB, ARRAY_BYTES);
	hipMalloc((void**)&d_arrC, ARRAY_BYTES);

	hipMemcpy(d_arrA, h_arrA, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_arrB, h_arrB, ARRAY_BYTES, hipMemcpyHostToDevice);
	
	hipEventRecord(start, 0);
	VecAdd <<<1 ,ARRAY_SIZE>>>(d_arrA,d_arrB,d_arrC,ARRAY_SIZE);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(h_arrC, d_arrC, ARRAY_BYTES, hipMemcpyDeviceToHost);
	
	for (i = 0; i < ARRAY_SIZE; i++)
	{
		printf("%f + %f = %f \n", h_arrA[i], h_arrB[i], h_arrC[i]);
	}
	printf("The Time taken by GPU is : %f", time);
	hipFree(d_arrA);
	hipFree(d_arrB);
	hipFree(d_arrC);
}















