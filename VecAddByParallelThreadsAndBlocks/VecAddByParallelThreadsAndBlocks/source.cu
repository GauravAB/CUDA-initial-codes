#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include ""

#define MAX 65535



__global__ void vecAdd(float* arr_A,float* arr_B, float* arr_C, int SIZE)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	while (tid < SIZE)
	{
		arr_C[tid] = arr_A[tid] + arr_B[tid];
		tid = tid + blockDim.x * gridDim.x;
	}
}



void main(int argc, char **argv)
{
	hipEvent_t start, stop;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float time;

    const int ARRAY_SIZE = atoi(argv[1]);
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
	
	float h_a[MAX];
	float h_b[MAX];
	float h_c[MAX];

	float *dev_a, *dev_b, *dev_c;

	int i;
	float f = 0.0;

	for (i = 0; i < ARRAY_SIZE; i++)
	{
		h_a[i] = f * f;
		h_b[i] = f * f * f;
		f++;
	}

	hipMalloc((void**)&dev_a, ARRAY_BYTES);
	hipMalloc((void**)&dev_b, ARRAY_BYTES);
	hipMalloc((void**)&dev_c, ARRAY_BYTES);

	hipMemcpy(dev_a, h_a, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, h_b, ARRAY_BYTES, hipMemcpyHostToDevice);

	hipEventRecord(start,0);
	vecAdd <<< (ARRAY_SIZE+127)/128,128 >>>(dev_a,dev_b,dev_c,ARRAY_SIZE);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(h_c, dev_c, ARRAY_BYTES, hipMemcpyDeviceToHost);

	/*for (i = 0; i < ARRAY_SIZE; i++)
	{
		printf("%f + %f = %f \n", h_a[i], h_b[i], h_c[i]);
	}*/
	
	printf("Time taken for kernel execution %f\n", time);

}



