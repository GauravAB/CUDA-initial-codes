
#include <hip/hip_runtime.h>
#include <stdio.h>


int main(void)
{
	int dev;
	int no_dev;
	hipDeviceProp_t prop;
	hipGetDevice(&dev);
	hipGetDeviceCount(&no_dev);
	hipGetDeviceProperties(&prop, dev);
	printf("number of devices are %d\n", no_dev);
	printf("Id of current CUDA device %d\n", dev);
	printf("Number of maximum Threads per Block are: %d\n", prop.maxThreadsPerBlock);
	prop.major = 4;
	prop.minor = 0;

	hipChooseDevice(&dev, &prop);
	printf("Id of chosen device id with closest match %d\n", dev);
	hipSetDevice(dev);
}