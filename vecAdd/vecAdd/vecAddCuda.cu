#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>


__global__ void vecAdd(float *a, float *b, float *c, int N)
{
	int idx = threadIdx.x;
	if (idx < N)
	{
		c[idx] = a[idx] + b[idx];
	}

} 


void main(void)
{
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	float time;
	
	const int ARRAY_SIZE = 1000;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	float h_a[ARRAY_SIZE];
	float h_b[ARRAY_SIZE];
	float h_c[ARRAY_SIZE];
	float *dev_a;
	float *dev_b;
	float *dev_c; 
	int i;
	float f = 0.0;

	for (i = 0; i < ARRAY_SIZE; i++)
	{
		h_a[i] = f;
		h_b[i] = f * f;
		f++;
	}

	hipMalloc((void**)&dev_a, ARRAY_BYTES);
	hipMalloc((void**)&dev_b, ARRAY_BYTES);
	hipMalloc((void**)&dev_c, ARRAY_BYTES);

	hipMemcpy(dev_a, h_a, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, h_b, ARRAY_BYTES, hipMemcpyHostToDevice);
	
	hipEventRecord(start, 0);
	vecAdd <<< 1, ARRAY_SIZE >>> (dev_a, dev_b, dev_c, ARRAY_SIZE);
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);

	hipMemcpy(h_c, dev_c, ARRAY_BYTES, hipMemcpyDeviceToHost);

	for (i = 0; i < ARRAY_SIZE; i++)
	{
		printf("%f * %f = %f \n", h_a[i], h_b[i], h_c[i]);
	}
	printf("Time taken by the GPU is : %f\n", time);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipEventDestroy(start);
	hipEventDestroy(end);
}















